#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <string>
#include <regex>
#include <array>

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include ""

__constant__ char* target = "pole";
__constant__ int target_length = 4;

__global__ void part1(char* d_hashes, int* d_values, int* d_limits, char* d_checksums, int size) {
	int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (id >= size) return;
	int count[26] = { 0 };
	int order[5] = { 0 };

	//check number letters
	for (int i = d_limits[id]; i < d_limits[id + 1]; ++i)
		if (d_hashes[i] != '-')
			++count[d_hashes[i] - 'a'];

	//check most common letters
	for (int steps = 0; steps < 5; ++steps) {
		for (int i = 25; i >= 0; --i) {
			if (count[order[steps]] <= count[i]) {
				order[steps] = i;
			}
		}
		count[order[steps]] = -1;
	}

	//validate checksum
	for (int i = 0; i < 5; ++i) {
		if (order[i] + 'a' != d_checksums[(id * 5) + i]) {
			d_values[id] = 0;  //set to 0 to neutralize the decoy
			break;
		}
	}
}

__global__ void part2(char* d_hashes, int* d_values, int* d_limits, char* d_checksums, int size) {
	int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (id >= size) return;  //thread is out of bound
	if (d_values[id] == 0) return;  //this is a decoy

	//Apply cypher
	for (int i = d_limits[id]; i < d_limits[id + 1]; ++i) {
		d_hashes[i] = (((d_hashes[i] - 'a') + d_values[id]) % 26) + 'a';
	}

	//look for target word
	for (int i = d_limits[id]; i < d_limits[id + 1]; ++i) {
		if (d_limits[id + 1] > i + target_length) {
			int nbr_valid = 0;
			for (int j = 0; j < target_length; ++j) {
				if (d_hashes[i+j] == target[j]) {
					++nbr_valid;
				}
			}
			if (nbr_valid == target_length) return; //legit - keep the value
		}
	}
	d_values[id] = 0; //target isn't found, set value to zero
}

int main() {
	std::string hash_str{};
	std::string check_str{};
	thrust::host_vector<int> values{};
	thrust::host_vector<int> limits{};
	limits.push_back(0);

	{ //File manipulation
		std::ifstream file("input.txt");
		std::string line = "";
		std::regex reg("([a-z,-]+)-([0-9]+).([a-z]+).");
		std::smatch cm;

		while (std::getline(file, line)) {
			std::regex_match(line, cm, reg);
			values.push_back(std::stoi(cm[2]));
			hash_str += cm[1];
			check_str += cm[3];
			limits.push_back(hash_str.size());
		}
		file.close();
	}

	char* d_hashes = 0;
	char* d_checksums = 0;

	hipMalloc((void**)&d_hashes, sizeof(char) * hash_str.size());
	hipMemcpy(d_hashes, hash_str.data(), sizeof(char) * hash_str.size(), hipMemcpyHostToDevice);

	hipMalloc((void**)&d_checksums, sizeof(char) * check_str.size());
	hipMemcpy(d_checksums, check_str.data(), sizeof(char) * check_str.size(), hipMemcpyHostToDevice);

	thrust::device_vector<int> d_values = values;
	thrust::device_vector<int> d_limits = limits;

	//Neutralize decoys
	part1 << < 1 + (values.size() / 32), 32 >> > (d_hashes,
		thrust::raw_pointer_cast(d_values.data()),
		thrust::raw_pointer_cast(d_limits.data()),
		d_checksums,
		values.size());

	hipDeviceSynchronize();
	int sum = thrust::reduce(d_values.begin(), d_values.end());  //All the decoy's values are set to 0
	std::cout << "The sum of sectors is: " << std::to_string(sum) << std::endl;

	//Neutralize anything that does't contain the target word
	part2 << < values.size(), 1 >> > (d_hashes,
		thrust::raw_pointer_cast(d_values.data()),
		thrust::raw_pointer_cast(d_limits.data()),
		d_checksums,
		values.size());

	hipDeviceSynchronize();
	sum = thrust::reduce(d_values.begin(), d_values.end()); //Normally, only one value is left
	std::cout << "The north pole's sector ID is: " << std::to_string(sum) << std::endl;  

	std::cin.get();
	return 0;
}